#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "src/fastertransformer/kernels/llama/decoder_masked_groupedquery_attention.h"
#include "src/fastertransformer/kernels/llama/decoder_masked_groupedquery_attention/decoder_masked_groupedquery_attention_template.hpp"
#include "src/fastertransformer/kernels/decoder_masked_multihead_attention_utils.h"
#include "src/fastertransformer/utils/cuda_bf16_wrapper.h"
#include <assert.h>
#include <float.h>
#include <type_traits>

template<typename T, typename KERNEL_PARAMS_TYPE>
void groupedquery_attention_(const KERNEL_PARAMS_TYPE& params, const hipStream_t& stream)
{
    switch (params.hidden_size_per_head) {
        case 32:
            mgqa_launch_kernel<T, 32, 32, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 48:
            mgqa_launch_kernel<T, 48, 64, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 64:
            mgqa_launch_kernel<T, 64, 64, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 80:
            mgqa_launch_kernel<T, 80, 128, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 96:
            mgqa_launch_kernel<T, 96, 128, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 128:
            mgqa_launch_kernel<T, 128, 128, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 144:
            mgqa_launch_kernel<T, 144, 256, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 160:
            mgqa_launch_kernel<T, 160, 256, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 192:
            mgqa_launch_kernel<T, 192, 256, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 224:
            mgqa_launch_kernel<T, 224, 256, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        case 256:
            mgqa_launch_kernel<T, 256, 256, KERNEL_PARAMS_TYPE>(params, stream);
            break;
        default:
            assert(false);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void masked_groupedquery_attention(const Masked_groupedquery_attention_params<float>& params, const hipStream_t& stream)
{
    groupedquery_attention_<float, Masked_groupedquery_attention_params<float>>(params, stream);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void masked_groupedquery_attention(const Masked_groupedquery_attention_params<uint16_t>& params, const hipStream_t& stream)
{
    groupedquery_attention_<uint16_t, Masked_groupedquery_attention_params<uint16_t>>(params, stream);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef ENABLE_BF16
void masked_groupedquery_attention(const Masked_groupedquery_attention_params<__hip_bfloat16>& params,
                                const hipStream_t&                                     stream)
{
    groupedquery_attention_<__hip_bfloat16, Masked_groupedquery_attention_params<__hip_bfloat16>>(params, stream);
}
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef ENABLE_FP8
void masked_groupedquery_attention(const Masked_groupedquery_attention_params<__hip_fp8_e4m3_fnuz>& params,
                                const hipStream_t&                                     stream)
{
    groupedquery_attention_<__hip_fp8_e4m3_fnuz, Masked_groupedquery_attention_params<__hip_fp8_e4m3_fnuz>>(params, stream);
}
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////
