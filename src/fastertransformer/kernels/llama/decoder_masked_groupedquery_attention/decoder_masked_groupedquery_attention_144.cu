#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "decoder_masked_groupedquery_attention_template.hpp"
#include "src/fastertransformer/kernels/llama/decoder_masked_groupedquery_attention.h"
#include "src/fastertransformer/kernels/decoder_masked_multihead_attention_utils.h"
#include "src/fastertransformer/utils/cuda_bf16_wrapper.h"
#include <assert.h>
#include <float.h>
#include <type_traits>

////////////////////////////////////////////////////////////////////////////////////////////////////

#define MGQA_LAUNCH_KERNEL(                                                                                            \
    T, Dh, Dh_MAX, THDS_PER_KEY, THDS_PER_VALUE, THDS_PER_BLOCK, HAS_BEAMS, stream)                \
    size_t smem_sz = mmha::smem_size_in_bytes<T>(params, THDS_PER_VALUE, THDS_PER_BLOCK);          \
    dim3   grid(params.num_heads, params.batch_size);                                                                  \
    mmha::masked_groupedquery_attention_kernel<T,                                                                         \
                                            Dh,                                                                        \
                                            Dh_MAX,                                                                    \
                                            THDS_PER_KEY,                                                              \
                                            THDS_PER_VALUE,                                                            \
                                            THDS_PER_BLOCK,                                                            \
                                            HAS_BEAMS><<<grid, THDS_PER_BLOCK, smem_sz, stream>>>(params)

////////////////////////////////////////////////////////////////////////////////////////////////////

template<typename T, int Dh, int Dh_MAX, typename KERNEL_PARAMS_TYPE>
void mgqa_launch_kernel(const KERNEL_PARAMS_TYPE& params, const hipStream_t& stream)
{
    constexpr int  THREADS_PER_VALUE  = Dh_MAX * sizeof(T) / 16;
    int            tlength            = params.timestep;
    if (params.cache_indir == nullptr) {
        if (tlength < 32) {
            MGQA_LAUNCH_KERNEL(T, Dh, Dh_MAX, 4, THREADS_PER_VALUE, 64, false, stream);
        }
        else if (tlength < 2048) {
            MGQA_LAUNCH_KERNEL(T, Dh, Dh_MAX, 2, THREADS_PER_VALUE, 128, false, stream);
        }
        else {
            MGQA_LAUNCH_KERNEL(T, Dh, Dh_MAX, 1, THREADS_PER_VALUE, 256, false, stream);
        }
    }
    else {
        if (tlength < 32) {
            MGQA_LAUNCH_KERNEL(T, Dh, Dh_MAX, 4, THREADS_PER_VALUE, 64, true, stream);
        }
        else if (tlength < 2048) {
            MGQA_LAUNCH_KERNEL(T, Dh, Dh_MAX, 2, THREADS_PER_VALUE, 128, true, stream);
        }
        else {
            MGQA_LAUNCH_KERNEL(T, Dh, Dh_MAX, 1, THREADS_PER_VALUE, 256, true, stream);
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template void mgqa_launch_kernel<float, 144, 256, GroupedQuery_attention_params<float>>(
    const GroupedQuery_attention_params<float>& params, const hipStream_t& stream);
template void mgqa_launch_kernel<uint16_t, 144, 256, GroupedQuery_attention_params<uint16_t>>(
    const GroupedQuery_attention_params<uint16_t>& params, const hipStream_t& stream);
#ifdef ENABLE_BF16
template void mgqa_launch_kernel<__hip_bfloat16, 144, 256, GroupedQuery_attention_params<__hip_bfloat16>>(
    const GroupedQuery_attention_params<__hip_bfloat16>& params, const hipStream_t& stream);
#endif
#ifdef ENABLE_FP8
template void mgqa_launch_kernel<__hip_fp8_e4m3_fnuz, 144, 256, GroupedQuery_attention_params<__hip_fp8_e4m3_fnuz>>(
    const GroupedQuery_attention_params<__hip_fp8_e4m3_fnuz>& params, const hipStream_t& stream);
#endif

#undef MGQA_LAUNCH_KERNEL
